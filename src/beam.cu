#include "hip/hip_runtime.h"


#include <math.h>
#include <stdio.h>

// Thread block size
#define BLOCK_SIZE 256

extern "C" void cudaBeamWrapper(int *res, const int *first, const int *last, int n_bytes);

// // Allocates an array with random integer entries.
// void randomInit(int *data, int size)
// {
//     for (int i = 0; i < size; ++i)
//         data[i] = rand();
// }

// __global__ void deviceDiffKernel(int *in_1, int *in_2, int *out, int N)
// {

//     int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y + 1;

//     out[idy * N + idx] = fabs((double)(in_1[idy * N + idx] - in_2[idy * N + idx]));
// }

// __global__ void r_vecKernel(double *phi, double *theta, int *out, int n_bytes, int N) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < N)
//     {
//         float *cosVal;
//         float *sinsinVal;
//         float *sincosVal;
//         out[i] = {sincosVal, sinsinVal, cosVal};
//         sinsinVal = sinf((double)(&theta)) * sinf((double)(&phi));
//     }
// }

// // Manage Thread Divergence
// __global__ void reductionKernel(int *input, int *results, int n) 
// {
//     extern __shared__ int sdata[];
//     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//     unsigned int tx = threadIdx.x;

//     // load input into __shared__ memory
//     int x = INT_MIN;
//     if (i < n)
//         x = input[i];
//     sdata[tx] = x;
//     __syncthreads();

//     // block-wide reduction
//     for (unsigned int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
//     {
//         __syncthreads();
//         if (tx < offset)
//         {
//             if (sdata[tx + offset] > sdata[tx])
//                 sdata[tx] = sdata[tx + offset];
//         }
//     }

//     // finally, thread 0 writes the result
//     if (threadIdx.x == 0)
//     {
//         // the result is per-block
//         results[blockIdx.x] = sdata[0];
//     }
// }

//     int r_vec(double phi, double theta, int n_bytes)
// {
//     int r[3];
//     double cosVal;
//     double sinVal;
//     double sincosVal;
//     // sincos(twopit * f, &sinVal, &cosVal);

//     return 0
// }

__global__ void beamKernel(int *res, const int *a, const int *b, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        res[i] = a[i] * b[i];
    }
}

// Cuda Wrapper for `beamKernel` used by C or Cython code
void cudaBeamWrapper(int *res, const int *first, const int *last, int n_bytes)
{
    // Setup buffers for GPU
    int *dev_res = nullptr;
    int *dev_first = nullptr;
    int *dev_last = nullptr;

    // Allocate memory on GPU for three vectors
    hipMalloc((void **)&dev_res, n_bytes * sizeof(int));
    hipMalloc((void **)&dev_first, n_bytes * sizeof(int));
    hipMalloc((void **)&dev_last, n_bytes * sizeof(int));

    // Copy allocated host memory to device
    hipMemcpy(dev_first, first, n_bytes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_last, last, n_bytes * sizeof(int), hipMemcpyHostToDevice);

    // Compute the result using one thread per element in vector
    // 2 is number of computational blocks and (n_bytes + 1) / 2 is a number of threads in a block
    beamKernel<<<2, (n_bytes + 1) / 2>>>(dev_res, dev_first, dev_last, n_bytes);

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(res, dev_res, n_bytes * sizeof(int), hipMemcpyDeviceToHost);

    // Release allocated memory
    hipFree(dev_res);
    hipFree(dev_first);
    hipFree(dev_last);

    hipDeviceReset();
}



// int main(int argc, char **argv)
// {
//     const int arraySize = BYTES;
//     int res[arraySize] = {0};
//     int first[arraySize];
//     int last[arraySize];

//     // Inititate random values
//     int i;
//     for (i = 0; i < BYTES; i++)
//     {
//         first[i] = rand();
//     }
//     for (i = 0; i < BYTES; i++)
//     {
//         last[i] = rand();
//     }
    
//     cudaBeamWrapper(res, first, last, arraySize);
//     int loop;
//     for (loop = 0; loop < BYTES; loop++)
//         printf("%d ", res[loop]);
//     printf("\n");
//     cudaDeviceReset();

//     return 0;
// }